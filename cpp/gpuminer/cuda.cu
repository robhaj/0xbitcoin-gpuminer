#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>
#include <string.h>
#include <map>

#ifndef _WIN32
#include <unistd.h>
#endif
// include thrust
#ifndef __cplusplus
#include <thrust/version.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#else
#include <ctype.h>
#endif

#include "miner.h"

#include "hip/hip_runtime.h"
hipStream_t gpustream[MAX_GPUS] = { 0 };
int opt_api_listen;

// CUDA Devices on the System
extern "C" int cuda_num_devices()
{
	int version;
	hipError_t err = hipDriverGetVersion(&version);
	if (err != hipSuccess)
	{
		exit(1);
	}

	int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
	if (maj < 5 || (maj == 5 && min < 5))
	{
		exit(1);
	}

	int GPU_N;
	err = hipGetDeviceCount(&GPU_N);
	if (err != hipSuccess)
	{
		exit(1);
	}
	return GPU_N;
}

extern "C" void cuda_devicenames()
{
	hipError_t err;
	int GPU_N;
	err = hipGetDeviceCount(&GPU_N);
	if (err != hipSuccess)
	{
		exit(1);
	}

	for (int i = 0; i < GPU_N*opt_n_gputhreads; i++)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, device_map[i / opt_n_gputhreads]);

		device_name[i] = strdup(props.name);
		device_sm[i] = (props.major * 100 + props.minor * 10);
	}
}

// Can't be called directly in cpu-miner.c
extern "C" void cuda_devicereset()
{
	hipDeviceSynchronize();
	hipDeviceReset();
}


extern "C" void cuda_print_devices()
{
	int ngpus = cuda_num_devices();
	cuda_devicenames();
	for (int n = 0; n < ngpus; n++) {
		int m = device_map[n % MAX_GPUS];
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, m);
		if (!opt_n_threads || n < opt_n_threads) {
			fprintf(stderr, "GPU #%d: SM %d.%d %s\n", m, props.major, props.minor, device_name[n]);
		}
	}
}

extern "C" static bool substringsearch(const char *haystack, const char *needle, int &match)
{
	int hlen = (int) strlen(haystack);
	int nlen = (int) strlen(needle);
	for (int i=0; i < hlen; ++i)
	{
		if (haystack[i] == ' ') continue;
		int j=0, x = 0;
		while(j < nlen)
		{
			if (haystack[i+x] == ' ') {++x; continue;}
			if (needle[j] == ' ') {++j; continue;}
			if (needle[j] == '#') return ++match == needle[j+1]-'0';
			if (tolower(haystack[i+x]) != tolower(needle[j])) break;
			++j; ++x;
		}
		if (j == nlen) return true;
	}
	return false;
}

// CUDA Gerät nach Namen finden (gibt Geräte-Index zurück oder -1)
extern "C" int cuda_finddevice(char *name)
{
	int num = cuda_num_devices();
	int match = 0;
	for (int i=0; i < num; ++i)
	{
		hipDeviceProp_t props;
		if (hipGetDeviceProperties(&props, i) == hipSuccess)
			if (substringsearch(props.name, name, match)) return i;
	}
	return -1;
}

extern "C" uint32_t device_intensity(int thr_id, const char *func, uint32_t defcount)
{
	uint32_t throughput = gpus_intensity[thr_id] ? gpus_intensity[thr_id] : defcount;
	if(opt_api_listen!=0) api_set_throughput(thr_id, throughput);
	return throughput;
}

// Zeitsynchronisations-Routine von cudaminer mit CPU sleep
typedef struct { double value[8]; } tsumarray;

extern "C" hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
	hipError_t result = hipSuccess;
	if (situation >= 0)
	{
		static std::map<int, tsumarray> tsum;
		double tsync = 0.0;
		double tsleep = 0.95;

		double a = 0.95, b = 0.05;
		if (tsum.find(situation) == tsum.end()) { a = 0.5; b = 0.5; } // faster initial convergence
		tsleep = 0.95*tsum[situation].value[thr_id];
		if (hipStreamQuery(stream) == hipErrorNotReady)
		{
			usleep((useconds_t)(1e6*tsleep));
			struct timeval tv_start, tv_end;
			gettimeofday(&tv_start, NULL);
			result = hipStreamSynchronize(stream);
			gettimeofday(&tv_end, NULL);
			tsync = 1e-6 * (tv_end.tv_usec - tv_start.tv_usec) + (tv_end.tv_sec - tv_start.tv_sec);
		}
		if (tsync >= 0) tsum[situation].value[thr_id] = a * tsum[situation].value[thr_id] + b * (tsleep + tsync);
	}
	else
		result = hipStreamSynchronize(stream);
	return result;
}


extern "C" int cuda_gpu_clocks(struct cgpu_info *gpu)
{
	hipDeviceProp_t props;
	if (hipGetDeviceProperties(&props, gpu->gpu_id) == hipSuccess) {
		gpu->gpu_clock = props.clockRate;
		gpu->gpu_memclock = props.memoryClockRate;
		gpu->gpu_mem = props.totalGlobalMem;
		return 0;
	}
	return -1;
}

extern "C" void cudaReportHardwareFailure(int thr_id, hipError_t err, const char* func)
{
	struct cgpu_info *gpu = &thr_info[thr_id].gpu;
	gpu->hw_errors++;
	sleep(1);
}
